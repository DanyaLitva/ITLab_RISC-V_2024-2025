#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <ctime>
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

using float16_t = half;

const int manLength = 10;
const int expLength = 5;
const int signLength = 1;
const int shiftExp = (1 << (expLength - 1)) - 1;

void print_float16_bites(float16_t f) {
    uint8_t bytes[2];
    bool bites[16];
    for (int i = 0; i < sizeof(f); i++) bytes[i] = *((uint8_t*)(&f) + i);
    for (int i = 0; i < 16; ++i) bites[15 - i] = (bytes[i / 8] >> (i % 8)) & 1;
    for (int i = 0; i < 16; ++i) {
        if (i == 1 || i == 6) std::cout << "_";
        std::cout << bites[i];
    }
}

unsigned short float16_bites_to_short(float16_t f) {
    unsigned short temp = 0;
    uint8_t bytes[2];
    bool bites[16];
    for (int i = 0; i < sizeof(f); i++) bytes[i] = *((uint8_t*)(&f) + i);
    for (int i = 0; i < 16; ++i) bites[15 - i] = (bytes[i / 8] >> (i % 8)) & 1;
    for (int i = 0; i < 16; ++i) temp |= (bites[15 - i] << i);
    return temp;
}

class FP16 {
private:
    unsigned int man : manLength;
    unsigned int exp : expLength;
    unsigned int sign : signLength;

public:
    FP16(int _sign = 0, int _exp = 0, int _man = 0) :sign(_sign), exp(_exp), man(_man) {}

    FP16(float16_t f) {
        unsigned short bits = float16_bites_to_short(f);
        sign = (bits >> 15) & 0x1;
        exp = (bits >> manLength) & ((1 << expLength) - 1);
        man = bits & ((1 << manLength) - 1);
    }

    bool IsSubnormal() { return (exp == 0) && (man != 0); }
    bool IsInf() { return exp == ((1 << expLength) - 1) && (man == 0); }
    bool IsNull() { return (exp == 0) && (man == 0); }
    bool IsNan() { return ((exp == ((1 << expLength) - 1)) && (man != 0)); }
    void printFP16_bites() {
        std::cout << sign << "_";
        for (char i = 0; i < expLength; ++i) std::cout << ((exp >> (expLength - i - 1)) & 1);
        std::cout << "_";
        for (char i = 0; i < manLength; ++i) std::cout << ((man >> (manLength - i - 1)) & 1);
    }

    operator float16_t() const {
        unsigned short bits = (sign << 15) | (exp << manLength) | man;
        return *reinterpret_cast<float16_t*>(&bits);
    }

    FP16& operator=(FP16& N) {
        if (this != &N) {
            sign = N.sign;
            exp = N.exp;
            man = N.man;
        }
        return *this;
    }

    uint16_t get_int() {
        uint16_t temp = (sign << (manLength + expLength)) + (exp << manLength) + man;
        return temp;
    }

    FP16 operator+(FP16 right) {
        if (IsNull()) return right;
        if (right.IsNull()) return *this;

        FP16 Res;
        uint16_t temp = 0;
        int8_t diff = exp - right.exp;
        int16_t shift;
        uint8_t max_exp;
        uint32_t lost_bit = 0;

        if (diff < 0) max_exp = right.exp;
        else max_exp = exp;



        if ((exp == right.exp)) {
            shift = 1 << (manLength);
            if (man >= right.man) {
                temp = (1 << manLength) + man + (1 - int(2 * ((this->sign + right.sign) % 2))) * (shift + right.man);
                Res.sign = sign;
            }
            else {
                temp = (1 << manLength) + right.man + (1 - int(2 * ((this->sign + right.sign) % 2))) * (man + shift);
                Res.sign = right.sign;
            }
        }

        if (this->exp > right.exp) {
            shift = 1 << (manLength - diff);
            if ((manLength - diff) < 0) shift = 0;
            temp = (1 << manLength) + this->man + (1 - int(2 * ((this->sign + right.sign) % 2))) * (shift + (right.man >> diff));
            lost_bit = right.man & (((1 << (diff)) - 1));
            lost_bit += (1 << manLength) & (((1 << (diff)) - 1));
            Res.sign = sign;
        }

        if (this->exp < right.exp) {
            diff *= -1;
            shift = 1 << (manLength - diff);
            if (manLength - diff < 0) shift = 0;
            temp = (1 << manLength) + right.man + (1 - int(2 * ((this->sign + right.sign) % 2))) * (shift + (this->man >> diff));
            lost_bit = man & (((1 << (diff)) - 1));
            lost_bit += (1 << manLength) & (((1 << (diff)) - 1));
            Res.sign = right.sign;
        }

        if (IsSubnormal() || right.IsSubnormal()) {
            if (IsSubnormal() && right.IsSubnormal()) {
                if (man >= right.man) {
                    temp = man + (1 - (2 * ((this->sign + right.sign) % 2))) * right.man;
                    Res.sign = sign;
                }
                else {
                    temp = right.man + (1 - (2 * ((this->sign + right.sign) % 2))) * man;
                    Res.sign = right.sign;
                }
            }
            else {
                //left subnormal
                if (IsSubnormal()) {
                    diff = right.exp - 1;
                    temp = (1 << manLength) + right.man + (1 - (2 * ((this->sign + right.sign) % 2))) * (this->man >> diff);
                    lost_bit = man & (((1 << (diff)) - 1));
                    Res.sign = right.sign;
                    max_exp = right.exp;
                }
                //right subnormal
                else {
                    diff = exp - 1;
                    temp = (1 << manLength) + this->man + (1 - (2 * ((this->sign + right.sign) % 2))) * (right.man >> diff);
                    lost_bit = right.man & (((1 << (diff)) - 1));
                    Res.sign = sign;
                    max_exp = exp;
                }
            }
        }

        if (sign != right.sign && lost_bit != 0) {
            temp--;
            lost_bit = (1 << (diff)) - lost_bit;
        }


        while (temp < (1 << manLength) && max_exp>1) {
            temp <<= 1;
            max_exp--;
            temp += (lost_bit >> (diff - 1)) & 1;
            lost_bit = lost_bit & ((1 << (diff - 1)) - 1);
            diff--;
        }

        if (temp < (1 << manLength) && max_exp == 1) {
            max_exp--;
            temp += (lost_bit >> (diff - 1)) & 1;
            lost_bit = lost_bit & ((1 << (diff - 1)) - 1);
            diff--;
        }


        if (temp == 0) {
            Res.sign = 0;
            Res.exp = 0;
            Res.man = 0;
            return Res;
        }

        if (temp < (1 << manLength) && (max_exp == 1)) {
            temp += (1 << manLength);
            max_exp = 0;
        }

        if (temp >= (1 << (manLength + 1)) && (max_exp > 0)) {
            lost_bit += ((temp & 1) << (diff));
            diff++;
            temp = temp >> 1;
            max_exp++;
        }


        if ((lost_bit >> (diff - 1)) & 1) {
            lost_bit = lost_bit & ((1 << (diff - 1)) - 1);
            if (lost_bit > 0) {
                temp++;
            }
            else {
                temp += (temp & 1);
            }
            if ((temp >= (1 << (manLength + 1))) && (max_exp >= 1)) {
                temp = temp >> 1;
                max_exp++;
            }
        }


        if (temp < (1 << manLength) && max_exp == 1) {
            max_exp = 0;
        }

        if (temp >= (1 << manLength) && (max_exp == 0)) {
            max_exp = 1;
            temp -= (1 << manLength);
        }

        if ((max_exp >= ((1 << expLength) - 1)) || IsInf() || right.IsInf() || IsNan() || right.IsNan()) {
            //inf +- inf
            if (IsInf() && right.IsInf()) {
                if (sign != right.sign) {
                    Res.exp = (1 << expLength);
                    Res.man = 1;
                    return Res;
                }
                else {
                    return *this;
                }
            }

            if (IsInf()) return *this;

            if (right.IsInf()) return right;

            if (IsNan()) return *this;

            if (right.IsNan()) return right;

            //if result is inf
            if (max_exp >= ((1 << expLength) - 1)) {
                Res.exp = ((1 << expLength) - 1);
                Res.man = 0;
                return Res;
            }
        }


        Res.exp = max_exp;
        Res.man = (temp - (1 << manLength));
        if (max_exp == 0) Res.man = temp;
        return Res;
    }

    FP16 operator-(FP16 right) {
        right.sign += 1;
        return *this + right;
    }

    FP16 operator*(FP16 right) {
        FP16 Res;
        Res.sign = sign ^ right.sign;
        uint16_t temp_man;
        int16_t temp_exp;
        uint32_t lost_bit;
        uint8_t diff = manLength;
        if (IsNull() || right.IsNull()) {
            Res.exp = 0;
            Res.man = 0;
            return Res;
        }
        if (!IsSubnormal() && !right.IsSubnormal()) {
            temp_exp = exp + right.exp - shiftExp - shiftExp;
            temp_man = (1 << manLength) + man + right.man + uint16_t((uint32_t(man) * right.man) >> manLength);
        }
        else {
            if (IsSubnormal() && right.IsSubnormal()) {
                Res.exp = 0;
                Res.man = 0;
                return Res;
            }
            else {
                temp_exp = exp - shiftExp + right.exp - shiftExp + 1;
                if (IsSubnormal()) {
                    temp_man = man + uint16_t((uint32_t(man) * right.man) >> manLength);
                }
                else {
                    temp_man = right.man + uint16_t((uint32_t(man) * right.man) >> manLength);
                }
            }
        }

        lost_bit = (uint32_t(man) * right.man) & ((1 << manLength) - 1);

        if (temp_exp < -(shiftExp - 1)) {
            while (temp_exp < -(shiftExp - 1)) {
                lost_bit += (temp_man & 1) << diff;
                diff++;
                temp_man = temp_man >> 1;
                temp_exp++;
            }
            if (temp_man < (1 << manLength)) temp_exp = -shiftExp;
        }

        while (temp_man < (1 << manLength) && temp_exp>-(shiftExp - 1)) {
            temp_man <<= 1;
            temp_exp--;

            temp_man += (lost_bit >> (diff - 1)) & 1;
            lost_bit = lost_bit & ((1 << (diff - 1)) - 1);
            diff--;
        }

        if ((temp_man >= (1 << (manLength + 1))) && (temp_exp >= -(shiftExp - 1))) {
            lost_bit += ((temp_man & 1) << (diff));
            diff++;
            temp_man = temp_man >> 1;
            temp_exp++;
        }

        if ((lost_bit >> (diff - 1)) & 1) {
            lost_bit = lost_bit & ((1 << (diff - 1)) - 1);
            if (lost_bit > 0) {
                temp_man++;
            }
            else {
                temp_man += (temp_man & 1);
            }
            if ((temp_man >= (1 << (manLength + 1))) && (temp_exp >= -(shiftExp - 1))) {
                temp_man = temp_man >> 1;
                temp_exp++;
            }
        }

        if (temp_exp == -shiftExp && temp_man >= (1 << manLength)) {
            temp_exp++;
        }

        if ((temp_exp >= ((1 << expLength) - 1 - shiftExp)) || IsInf() || right.IsInf() || IsNan() || right.IsNan()) {
            //inf +- inf
            if (IsInf() && right.IsInf()) {
                if (sign != right.sign) {
                    Res.exp = (1 << expLength);
                    Res.man = 1;
                    return Res;
                }
                else {
                    return *this;
                }
            }

            if (IsInf()) return *this;

            if (right.IsInf()) return right;

            if (IsNan()) return *this;

            if (right.IsNan()) return right;

            //if result is inf
            if (temp_exp >= (((1 << expLength) - 1 - shiftExp))) {
                Res.exp = (1 << expLength) - 1;
                Res.man = 0;
                return Res;
            }
        }

        if ((temp_man < (1 << manLength)) && (temp_exp == -(shiftExp - 1))) {
            Res.exp = 0;
            Res.man = temp_man;
            return Res;
        }

        Res.exp = temp_exp + shiftExp;
        Res.man = temp_man - (1 << manLength);

        return Res;
    }

    friend FP16 fma(FP16 a, FP16 b, FP16 c);
};

//a*b + c
FP16 fma(FP16 a, FP16 b, FP16 c) {
    FP16 Res;
    uint32_t temp_man;


    return Res;
}



void print_bites(FP16 a) { a.printFP16_bites(); }
void print_bites(float16_t a) { print_float16_bites(a); }

void Test_Add();
void Test_Sub();
void Test_Mul();
void TimeTest();

using namespace std;
int main() {
    FP16 A(0, 0, 1);
    FP16 B(1, 17, 1);

    float16_t a = A;
    float16_t b = B;
    float16_t c;
    cout << "A: "; print_bites(A); cout << ", " << __half2float(A) << endl;
    cout << "B: "; print_bites(B); cout << ", " << __half2float(B) << endl;
    cout << "A + B: "; print_bites(A + B); cout << " - My" << ", " << __half2float(A + B) << endl;
    cout << "a + b: "; print_bites(a + b); cout << ", " << __half2float(a + b) << endl;
    cout << "A - B: "; print_bites(A - B); cout << " - My" << ", " << __half2float(A - B) << endl;
    cout << "a - b: "; print_bites(a - b); cout << ", " << __half2float(a - b) << endl;
    cout << "A * B: "; print_bites(A * B); cout << ", " << __half2float(A * B) << " - My" << endl;
    cout << "a * b: "; print_bites(a * b); cout << ", " << __half2float(a * b) << endl;
    
    Test_Mul();
    

    return 0;
}


void Test_Add() {
    cout << "Addition test:" << endl;
    float16_t a, b, c;
    for (size_t sign1 = 0; sign1 < 1; ++sign1) {
        for (size_t exp1 = 0; exp1 < (1 << expLength) - 1; ++exp1) {
            for (size_t man1 = 0; man1 < (1 << manLength); ++man1) {
                for (size_t sign2 = 0; sign2 < 1; ++sign2) {
                    for (size_t exp2 = 0; exp2 < (1 << expLength) - 1; ++exp2) {
                        for (size_t man2 = 0; man2 < (1 << manLength); ++man2) {
                            FP16 A(sign1, exp1, man1);
                            FP16 B(sign2, exp2, man2);
                            FP16 C = A + B;
                            a = A; b = B;
                            c = a + b;
                            if (abs(float16_bites_to_short(C) - float16_bites_to_short(c)) >= 1) {
                                cout << "A: ";
                                print_bites(A);
                                cout << endl;
                                cout << "B: ";
                                print_bites(B);
                                cout << endl;
                                cout << "A + B: ";
                                print_bites(A + B);
                                cout << " - My " << endl;
                                cout << "a + b: ";
                                print_bites(a + b);
                                cout << endl;
                                cout << sign1 << " " << exp1 << " " << man1 << " " << endl;
                                cout << sign2 << " " << exp2 << " " << man2 << " " << endl;
                                //return;
                            }
                        }
                    }
                }
            }
            cout << "Exp = " << exp1 << endl;
        }
    }
}

void Test_Sub() {
    cout << "Subrtact test:" << endl;
    float16_t a, b, c;
    for (size_t sign1 = 0; sign1 < 1; ++sign1) {
        for (size_t exp1 = 0; exp1 < (1 << expLength) - 1; ++exp1) {
            for (size_t man1 = 0; man1 < (1 << manLength); ++man1) {
                for (size_t sign2 = 0; sign2 < 1; ++sign2) {
                    for (size_t exp2 = 0; exp2 < (1 << expLength) - 1; ++exp2) {
                        for (size_t man2 = 0; man2 < (1 << manLength); ++man2) {
                            FP16 A(sign1, exp1, man1);
                            FP16 B(sign2, exp2, man2);
                            FP16 C = A - B;
                            a = A; b = B;
                            c = a - b;
                            if (abs(float16_bites_to_short(C) - float16_bites_to_short(c)) >= 1) {
                                cout << "A: ";
                                print_bites(A);
                                cout << endl;
                                cout << "B: ";
                                print_bites(B);
                                cout << endl;
                                cout << "A - B: ";
                                print_bites(A - B);
                                cout << " - My " << endl;
                                cout << "a - b: ";
                                print_bites(a - b);
                                cout << endl;
                                cout << sign1 << " " << exp1 << " " << man1 << " " << endl;
                                cout << sign2 << " " << exp2 << " " << man2 << " " << endl;
                                //return;

                            }
                        }
                    }
                }
            }
            cout << "Exp = " << exp1 << endl;
        }
    }
}


void Test_Mul() {
    cout << "Mult test:" << endl;
    float16_t a, b, c;
    for (size_t sign1 = 0; sign1 < 1; ++sign1) {
        for (size_t exp1 = 0; exp1 < (1 << expLength) - 1; ++exp1) {
            for (size_t man1 = 0; man1 < (1 << manLength); ++man1) {
                for (size_t sign2 = 0; sign2 < 1; ++sign2) {
                    for (size_t exp2 = 0; exp2 < (1 << expLength) - 1; ++exp2) {
                        for (size_t man2 = 0; man2 < (1 << manLength); ++man2) {
                            FP16 A(sign1, exp1, man1);
                            FP16 B(sign2, exp2, man2);
                            FP16 C = A * B;
                            a = A; b = B;
                            c = a * b;
                            if (abs(float16_bites_to_short(C) - float16_bites_to_short(c)) >= 1) {
                                cout << "A: ";
                                print_bites(A);
                                cout << endl;
                                cout << "B: ";
                                print_bites(B);
                                cout << endl;
                                cout << "A * B: ";
                                print_bites(A * B);
                                cout << " - My " << endl;
                                cout << "a * b: ";
                                print_bites(a * b);
                                cout << endl;
                                cout << sign1 << " " << exp1 << " " << man1 << " " << endl;
                                cout << sign2 << " " << exp2 << " " << man2 << " " << endl;
                                //return;                                
                            }
                        }
                    }
                }
            }
            cout << "Exp = " << exp1 << endl;
        }
    }
}

void TimeTest() {
    FP16 A;
    FP16 B;
    FP16 C;
    int start,end,t;
    cout << endl << "Addition Time Test:" << endl;
    start = clock();
    for (size_t sign1 = 0; sign1 < 1; ++sign1) {
        for (size_t exp1 = 0; exp1 < (1 << expLength) - 1; ++exp1) {
            for (size_t man1 = 0; man1 < (1 << manLength); ++man1) {
                for (size_t sign2 = 0; sign2 < 1; ++sign2) {
                    for (size_t exp2 = 0; exp2 < (1 << expLength) - 1; ++exp2) {
                        for (size_t man2 = 0; man2 < (1 << manLength); ++man2) {
                            A = FP16(sign1, exp1, man1);
                            B = FP16(sign2, exp2, man2);
                            C = A + B;
                        }
                    }
                }
            }
        }
    }
    end = clock();
    t = (end - start) / CLOCKS_PER_SEC;

    cout << "Time: " << t << " seconds" << endl;

    cout << endl << "Subtract Time Test:" << endl;

    start = clock();
    for (size_t sign1 = 0; sign1 < 1; ++sign1) {
        for (size_t exp1 = 0; exp1 < (1 << expLength) - 1; ++exp1) {
            for (size_t man1 = 0; man1 < (1 << manLength); ++man1) {
                for (size_t sign2 = 0; sign2 < 1; ++sign2) {
                    for (size_t exp2 = 0; exp2 < (1 << expLength) - 1; ++exp2) {
                        for (size_t man2 = 0; man2 < (1 << manLength); ++man2) {
                            A = FP16(sign1, exp1, man1);
                            B = FP16(sign2, exp2, man2);
                            C = A - B;
                        }
                    }
                }
            }
        }
    }
    end = clock();
    t = (end - start) / CLOCKS_PER_SEC;

    cout << "Time: " << t << " seconds" << endl;

    cout << endl << "Multiplication Time Test:" << endl;

    start = clock();
    for (size_t sign1 = 0; sign1 < 1; ++sign1) {
        for (size_t exp1 = 0; exp1 < (1 << expLength) - 1; ++exp1) {
            for (size_t man1 = 0; man1 < (1 << manLength); ++man1) {
                for (size_t sign2 = 0; sign2 < 1; ++sign2) {
                    for (size_t exp2 = 0; exp2 < (1 << expLength) - 1; ++exp2) {
                        for (size_t man2 = 0; man2 < (1 << manLength); ++man2) {
                            A = FP16(sign1, exp1, man1);
                            B = FP16(sign2, exp2, man2);
                            C = A * B;
                        }
                    }
                }
            }
        }
    }
    end = clock();
    t = (end - start) / CLOCKS_PER_SEC;

    cout << "Time: " << t << " seconds" << endl;

}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
